// gemm/kernels_cublas.cu
// cuBLAS SGEMM reference (row-major via swapped args): C = A*B

#include "kernels.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>

void launch_cublas(const float* A, const float* B, float* C, int M, int N, int K, hipStream_t s) {
  static hipblasHandle_t handle = nullptr;
  if (!handle) hipblasCreate(&handle);
  hipblasSetStream(handle, s);

  // Simple GEMM with scalar transforms
  const float alpha = 1.f, beta = 0.f;

  // cuBLAS expects matrices in col-major form but our code uses row-major.
  // We can simply pass the same matrices but with column leading dimensions to get desired results
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              N, M, K,
              &alpha,
              B, N,
              A, K,
              &beta,
              C, N);
}